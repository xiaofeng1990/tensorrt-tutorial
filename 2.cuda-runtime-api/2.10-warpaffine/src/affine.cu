
#include <hip/hip_runtime.h>
#include <stdio.h>
#define min(a, b) ((a) < (b) ? (a) : (b))
#define num_threads 512

typedef unsigned char uint8_t;

struct Size
{
    int width = 0, height = 0;

    Size() = default;
    Size(int w, int h)
        : width(w), height(h) {}
};

// 计算仿射变换矩阵
// 计算的矩阵是居中缩放
struct AffineMatrix
{
    /*
    建议先阅读代码，若有疑问，可点击抖音短视频进行辅助讲解(建议1.5倍速观看)
        - https://v.douyin.com/Nhr5UdL/
     */

    float i2d[6]; // image to dst(network), 2x3 matrix
    float d2i[6]; // dst to image, 2x3 matrix

    // 这里其实是求解imat的逆矩阵，由于这个3x3矩阵的第三行是确定的0, 0, 1，因此可以简写如下
    void invertAffineTransform(float imat[6], float omat[6])
    {
        float i00 = imat[0];
        float i01 = imat[1];
        float i02 = imat[2];
        float i10 = imat[3];
        float i11 = imat[4];
        float i12 = imat[5];

        // 计算行列式
        float D = i00 * i11 - i01 * i10;
        D = D != 0 ? 1.0 / D : 0;

        // 计算剩余的伴随矩阵除以行列式
        float A11 = i11 * D;
        float A22 = i00 * D;
        float A12 = -i01 * D;
        float A21 = -i10 * D;
        float b1 = -A11 * i02 - A12 * i12;
        float b2 = -A21 * i02 - A22 * i12;
        omat[0] = A11;
        omat[1] = A12;
        omat[2] = b1;
        omat[3] = A21;
        omat[4] = A22;
        omat[5] = b2;
    }

    void compute(const Size &from, const Size &to)
    {
        float scale_x = to.width / (float)from.width;
        float scale_y = to.height / (float)from.height;

        // 这里取min的理由是
        // 1. M矩阵是 from * M = to的方式进行映射，因此scale的分母一定是from
        // 2. 取最小，即根据宽高比，算出最小的比例，如果取最大，则势必有一部分超出图像范围而被裁剪掉，这不是我们要的
        // **
        float scale = min(scale_x, scale_y); // 缩放比例辅助视频讲解 https://v.douyin.com/NhrH8Gm/
        /**
        这里的仿射变换矩阵实质上是2x3的矩阵，具体实现是
        scale, 0, -scale * from.width * 0.5 + to.width * 0.5
        0, scale, -scale * from.height * 0.5 + to.height * 0.5

        这里可以想象成，是经历过缩放、平移、平移三次变换后的组合，M = TPS
        例如第一个S矩阵，定义为把输入的from图像，等比缩放scale倍，到to尺度下
        S = [
        scale,     0,      0
        0,     scale,      0
        0,         0,      1
        ]

        P矩阵定义为第一次平移变换矩阵，将图像的原点，从左上角，移动到缩放(scale)后图像的中心上
        P = [
        1,        0,      -scale * from.width * 0.5
        0,        1,      -scale * from.height * 0.5
        0,        0,                1
        ]

        T矩阵定义为第二次平移变换矩阵，将图像从原点移动到目标（to）图的中心上
        T = [
        1,        0,      to.width * 0.5,
        0,        1,      to.height * 0.5,
        0,        0,            1
        ]

        通过将3个矩阵顺序乘起来，即可得到下面的表达式：
        M = [
        scale,    0,     -scale * from.width * 0.5 + to.width * 0.5
        0,     scale,    -scale * from.height * 0.5 + to.height * 0.5
        0,        0,                     1
        ]
        去掉第三行就得到opencv需要的输入2x3矩阵
        **/

        /*
            + scale * 0.5 - 0.5 的主要原因是使得中心更加对齐，下采样不明显，但是上采样时就比较明显
            参考：https://www.iteye.com/blog/handspeaker-1545126
        */
        i2d[0] = scale;
        i2d[1] = 0;
        i2d[2] =
            -scale * from.width * 0.5 + to.width * 0.5 + scale * 0.5 - 0.5;

        i2d[3] = 0;
        i2d[4] = scale;
        i2d[5] =
            -scale * from.height * 0.5 + to.height * 0.5 + scale * 0.5 - 0.5;

        invertAffineTransform(i2d, d2i);
    }
};

__device__ void affine_project(float *matrix, int x, int y, float *proj_x, float *proj_y)
{

    // matrix
    // m0, m1, m2
    // m3, m4, m5
    *proj_x = matrix[0] * x + matrix[1] * y + matrix[2];
    *proj_y = matrix[3] * x + matrix[4] * y + matrix[5];
}

__global__ void warp_affine_bilinear_kernel(
    uint8_t *src, int src_line_size, int src_width, int src_height,
    uint8_t *dst, int dst_line_size, int dst_width, int dst_height,
    uint8_t fill_value, AffineMatrix matrix)
{
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    if (dx >= dst_width || dy >= dst_height)
        return;

    float c0 = fill_value, c1 = fill_value, c2 = fill_value;
    float src_x = 0;
    float src_y = 0;
    affine_project(matrix.d2i, dx, dy, &src_x, &src_y);

    /*
    建议先阅读代码，若有疑问，可点击抖音短视频进行辅助讲解(建议1.5倍速观看)
        - 双线性理论讲解：https://v.douyin.com/NhrH2tb/
        - 代码代码：https://v.douyin.com/NhrBqpc/
     */
    if (src_x < -1 || src_x >= src_width || src_y < -1 || src_y >= src_height)
    {
        // out of range
        // src_x < -1时，其高位high_x < 0，超出范围
        // src_x >= -1时，其高位high_x >= 0，存在取值
    }
    else
    {
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_values[] = {fill_value, fill_value, fill_value};
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t *v1 = const_values;
        uint8_t *v2 = const_values;
        uint8_t *v3 = const_values;
        uint8_t *v4 = const_values;
        if (y_low >= 0)
        {
            if (x_low >= 0)
                v1 = src + y_low * src_line_size + x_low * 3;

            if (x_high < src_width)
                v2 = src + y_low * src_line_size + x_high * 3;
        }

        if (y_high < src_height)
        {
            if (x_low >= 0)
                v3 = src + y_high * src_line_size + x_low * 3;

            if (x_high < src_width)
                v4 = src + y_high * src_line_size + x_high * 3;
        }

        c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
        c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
        c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
    }
    uint8_t *pdst = dst + dy * dst_line_size + dx * 3;
    pdst[0] = c0;
    pdst[1] = c1;
    pdst[2] = c2;
}

__global__ void warp_affine_bilinear_kernel_batch(
    uint8_t *src, int src_line_size, int src_fream_size, int src_width, int src_height,
    uint8_t *dst, int dst_line_size, int dst_fream_size, int dst_width, int dst_height,
    uint8_t batch_size, uint8_t fill_value, AffineMatrix matrix)
{
    int dx = blockDim.x * blockIdx.x + threadIdx.x;
    int dy = blockDim.y * blockIdx.y + threadIdx.y;
    int dz = threadIdx.z;
    int index = (((((blockIdx.z * gridDim.y) + blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.z + threadIdx.z) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    // printf("threadIdx.z = %d, blockIdx.z=%d, blockDim.z=%d\n", threadIdx.z, blockIdx.z, blockDim.z);
    // if (dz == 0)
    //     printf("dz %d \n", dz);
    if (dx >= dst_width || dy >= dst_height || dz >= batch_size)
        return;

    float c0 = fill_value, c1 = fill_value, c2 = fill_value;
    float src_x = 0;
    float src_y = 0;
    affine_project(matrix.d2i, dx, dy, &src_x, &src_y);

    uint8_t *src_index = src + src_fream_size * dz;
    // uint8_t *src_index = src;
    // uint8_t *dst_index = dst + dst_fream_size * dz;
    // uint8_t *dst_index = dst;
    // uint8_t *src_index = src + index;
    // uint8_t *src_index = src + src_fream_size * dz;
    uint8_t *dst_index = dst + dst_fream_size;
    /*
    建议先阅读代码，若有疑问，可点击抖音短视频进行辅助讲解(建议1.5倍速观看)
        - 双线性理论讲解：https://v.douyin.com/NhrH2tb/
        - 代码代码：https://v.douyin.com/NhrBqpc/
    */
    if (src_x < -1 || src_x >= src_width || src_y < -1 || src_y >= src_height)
    {
        // out of range
        // src_x < -1时，其高位high_x < 0，超出范围
        // src_x >= -1时，其高位high_x >= 0，存在取值
    }
    else
    {
        int y_low = floorf(src_y);
        int x_low = floorf(src_x);
        int y_high = y_low + 1;
        int x_high = x_low + 1;

        uint8_t const_values[] = {fill_value, fill_value, fill_value};
        float ly = src_y - y_low;
        float lx = src_x - x_low;
        float hy = 1 - ly;
        float hx = 1 - lx;
        float w1 = hy * hx, w2 = hy * lx, w3 = ly * hx, w4 = ly * lx;
        uint8_t *v1 = const_values;
        uint8_t *v2 = const_values;
        uint8_t *v3 = const_values;
        uint8_t *v4 = const_values;
        if (y_low >= 0)
        {
            if (x_low >= 0)
                v1 = src_index + y_low * src_line_size + x_low * 3;

            if (x_high < src_width)
                v2 = src_index + y_low * src_line_size + x_high * 3;
        }

        if (y_high < src_height)
        {
            if (x_low >= 0)
                v3 = src_index + y_high * src_line_size + x_low * 3;

            if (x_high < src_width)
                v4 = src_index + y_high * src_line_size + x_high * 3;
        }

        c0 = floorf(w1 * v1[0] + w2 * v2[0] + w3 * v3[0] + w4 * v4[0] + 0.5f);
        c1 = floorf(w1 * v1[1] + w2 * v2[1] + w3 * v3[1] + w4 * v4[1] + 0.5f);
        c2 = floorf(w1 * v1[2] + w2 * v2[2] + w3 * v3[2] + w4 * v4[2] + 0.5f);
    }
    uint8_t *pdst = dst_index + dy * dst_line_size + dx * 3;
    // uint8_t *pdst = dst + index;
    pdst[0] = c0;
    pdst[1] = c1;
    pdst[2] = c2;
}

void warp_affine_bilinear(
    uint8_t *src, int src_line_size, int src_width, int src_height,
    uint8_t *dst, int dst_line_size, int dst_width, int dst_height,
    uint8_t fill_value)
{
    dim3 block_size(32, 32); // blocksize 最大就是1024，这里用2d来看更好理解
    dim3 grid_size((dst_width + 31) / 32, (dst_height + 31) / 32);
    AffineMatrix affine;
    affine.compute(Size(src_width, src_height), Size(dst_width, dst_height));

    warp_affine_bilinear_kernel<<<grid_size, block_size, 0, nullptr>>>(
        src, src_line_size, src_width, src_height,
        dst, dst_line_size, dst_width, dst_height,
        fill_value, affine);
}

void warp_affine_bilinear_batch(
    uint8_t *src, int src_line_size, int src_frame_size, int src_width, int src_height,
    uint8_t *dst, int dst_line_size, int dst_frame_size, int dst_width, int dst_height,
    uint8_t batch_size, uint8_t fill_value)
{
    dim3 block_size(32, 32); // blocksize 最大就是1024，这里用2d来看更好理解
    dim3 grid_size((dst_width + 31) / 32, (dst_height + 31) / 32, batch_size);
    printf("grid size = %d * %d * %d \n", grid_size.x, grid_size.y, grid_size.z);
    printf("block size = %d * %d * %d \n", block_size.x, block_size.y, block_size.z);
    AffineMatrix affine;
    affine.compute(Size(src_width, src_height), Size(dst_width, dst_height));

    warp_affine_bilinear_kernel_batch<<<grid_size, block_size, 0, nullptr>>>(
        src, src_line_size, src_frame_size, src_width, src_height,
        dst, dst_line_size, dst_frame_size, dst_width, dst_height, batch_size,
        fill_value, affine);
}