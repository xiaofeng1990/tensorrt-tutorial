#include <stdio.h>
#include <hip/hip_runtime.h>


// CPU调用，在GPU上执行
__global__ void test_print_kernel(const float *pdata, int ndata)
{
    // threadIdx blockIdx  blockDim 内置变量

    // threadIdx;
    // blockIdx;
    // blockDim; threadIdx 表示block中thread索引
    // gridDim; blockIdx 表示grid中block的索引
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    int idy = threadIdx.y + blockIdx.y * blockDim.y;
    int idz = threadIdx.z + blockIdx.z * blockDim.z;
    /*
           dims           indexs
        gridDim.z  1     blockIdx.z  0
        gridDim.y  1     blockIdx.y  0
        gridDim.x  2     blockIdx.x  0-1
        blockDim.z 1     threadIdx.z 0
        blockDim.y 1     threadIdx.y 0
        blockDim.x 10    threadIdx.x 0-9

        Pseudo code:
        position = 0
        for i in 6:
            position *= dims[i]
            position += indexs[i]
    */
    // printf("Element[%d] = %f, threadIdx.x = %d, blockIdx.x=%d, blockDim.x=%d\n", idx, pdata[idx], threadIdx.x, blockIdx.x, blockDim.x);
    int index = (((((blockIdx.z * gridDim.y) + blockIdx.y) * gridDim.x + blockIdx.x) * blockDim.z + threadIdx.z) * blockDim.y + threadIdx.y) * blockDim.x + threadIdx.x;
    printf("Element[%d] = %f, \n", index, pdata[index]);
    printf("x = %d, y=%d, z=%d\n", idx, idy, idz);
}

void test_print(const float *pdata, int ndata)
{
    dim3 gridDim;
    dim3 blockDim;
    int nthreads = gridDim.x * gridDim.y * gridDim.z * blockDim.x * blockDim.y * blockDim.z;

    // gridDim(21亿, 65536, 65536) //
    // blockDim(1024, 64, 64)  blockDim.x * blockDim.y * blockDim.z<=1024
    //  <<< gridDim, blockDim,  bytes_of_shared_memory, stream>>>
    // gridDim 定义线程格尺寸里面有多少线程快 blockDim定义线程快里面有多少线程
    test_print_kernel<<<dim3(2, 2, 2), dim3(2, 2), 0, nullptr>>>(pdata, ndata);
    // 在核函数执行结束后，通过 cudaPeekAtLastError 判断是否执行错误
    //  cudaPeekAtLastError 和 cudaGetLastError 都可以获取错误代码
    //  cudaGetLastError 是获取错误代码后并清楚，也就是在执行一次 cudaGetLastError 获取到的会是 success
    //  而 cudaPeekAtLastError 是获取当前错误，但是再次执行 cudaPeekAtLastError 或者 cudaGetLastError 拿到的还是那个错
    //  cuda 的错误会传递，如果这里出错，不移除，那么后续的任意api的返回值都会是这个错误，都会失败

    hipError_t code = hipPeekAtLastError();
    if (code != hipSuccess)
    {
        const char *err_name = hipGetErrorName(code);
        const char *err_message = hipGetErrorString(code);
        printf("kernel error %s:%d  test_print_kernel failed. \n  code = %s, message = %s\n", __FILE__, __LINE__, err_name, err_message);
    }
}
